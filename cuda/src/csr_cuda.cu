#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "utils.h"

#define WARP_SIZE 32
#define BLOCK_SIZE 512  // Multiplo di 32

__global__ void csr_mult_warp_cuda_kernel(int num_rows, int *d_row_ptr, int *d_col_indices, 
                                     double *d_values, double *d_x, double *d_y) {
    __shared__ double sdata[BLOCK_SIZE];  
    __shared__ int ptrs[BLOCK_SIZE / WARP_SIZE][2];

    const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;  // ID globale del thread
    const int thread_in_warp = threadIdx.x & (WARP_SIZE - 1);     // ID del thread dentro il warp
    const int warp_id = thread_id / WARP_SIZE;                    // ID globale del warp
    const int warp_in_block = threadIdx.x / WARP_SIZE;            // Warp ID locale nel blocco
    const int num_warps = (BLOCK_SIZE / WARP_SIZE) * gridDim.x;   // Numero totale di warps

    // Ogni warp si occupa di una riga della matrice
    for (int row = warp_id; row < num_rows; row += num_warps) {
        // I primi due thread di ogni warp caricano gli indici di inizio/fine riga
        if (thread_in_warp < 2) {
            ptrs[warp_in_block][thread_in_warp] = d_row_ptr[row + thread_in_warp];
        }
        __syncthreads(); // Sincronizzazione per garantire che ptrs sia stato aggiornato

        int row_start = ptrs[warp_in_block][0];  
        int row_end = ptrs[warp_in_block][1];    

        // Calcolo del prodotto locale
        double sum = 0.0;
        for (int j = row_start + thread_in_warp; j < row_end; j += WARP_SIZE) {
            sum += d_values[j] * d_x[d_col_indices[j]];
        }

        // Riduzione all'interno del warp
        for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
            sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
        }

        // Il primo thread del warp scrive il risultato finale nella memoria globale
        if (thread_in_warp == 0) {
            d_y[row] = sum;
        }
    }
}


void cuda_csr_mult_warp(CSRMatrix *A, double *x, double *y, float *elapsed_time) {
    int *d_row_ptr, *d_col_indices;
    double *d_values, *d_x, *d_y;

    // Allocazione memoria sulla GPU
    hipMalloc((void **)&d_row_ptr, (A->rows + 1) * sizeof(int));
    hipMalloc((void **)&d_col_indices, A->nnz * sizeof(int));
    hipMalloc((void **)&d_values, A->nnz * sizeof(double));
    hipMalloc((void **)&d_x, A->cols * sizeof(double));
    hipMalloc((void **)&d_y, A->rows * sizeof(double));

    // Copia dati dalla CPU alla GPU
    hipMemcpy(d_row_ptr, A->row_ptr, (A->rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_indices, A->col_indices, A->nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values, A->values, A->nnz * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, A->cols * sizeof(double), hipMemcpyHostToDevice);

    // Configurazione e lancio del kernel CUDA 
    int num_warps = (A->rows + WARP_SIZE - 1) / WARP_SIZE;
    int num_blocks = (num_warps + (BLOCK_SIZE / WARP_SIZE) - 1) / (BLOCK_SIZE / WARP_SIZE);

    // Configurazione per il calcolo del tempo di esecuzione
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start, 0);

    csr_mult_warp_cuda_kernel<<<num_blocks, BLOCK_SIZE>>>(A->rows, d_row_ptr, d_col_indices, d_values, d_x, d_y);


    // registrazione del tempo di esecuzione
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Copia il risultato dalla GPU alla CPU
    hipMemcpy(y, d_y, A->rows * sizeof(double), hipMemcpyDeviceToHost);

    // allocazione del tempo
    hipEventElapsedTime(elapsed_time, start, stop);

    // Deallocazione della memoria sulla GPU
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_row_ptr);
    hipFree(d_col_indices);
    hipFree(d_values);
    hipFree(d_x);
    hipFree(d_y);
}


// Kernel CUDA per il prodotto matrice-vettore CSR
__global__ void csr_mult_cuda_kernel(int rows, int *d_row_ptr, int *d_col_indices, double *d_values, double *d_x, double *d_y) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;  // Ogni thread processa una riga

    if (row < rows) {
        double sum = 0.0;

        for (int j = d_row_ptr[row]; j < d_row_ptr[row + 1]; j++)
            sum += d_values[j] * d_x[d_col_indices[j]];
        
        d_y[row] = sum;
    }
}

void cuda_csr_mult(CSRMatrix *A, double *x, double *y, float *elapsed_time) {
    int *d_row_ptr, *d_col_indices;
    double *d_values, *d_x, *d_y;

    // Allocazione memoria sulla GPU
    hipMalloc((void **)&d_row_ptr, (A->rows + 1) * sizeof(int));
    hipMalloc((void **)&d_col_indices, A->nnz * sizeof(int));
    hipMalloc((void **)&d_values, A->nnz * sizeof(double));
    hipMalloc((void **)&d_x, A->cols * sizeof(double));
    hipMalloc((void **)&d_y, A->rows * sizeof(double));

    // Copia dati dalla CPU alla GPU
    hipMemcpy(d_row_ptr, A->row_ptr, (A->rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_indices, A->col_indices, A->nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values, A->values, A->nnz * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, A->cols * sizeof(double), hipMemcpyHostToDevice);

    // Configurazione e lancio del kernel CUDA
    int blockSize = 512;
    int gridSize = (A->rows + blockSize - 1) / blockSize;

    // Configurazione per il calcolo del tempo di esecuzione
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start, 0);

    csr_mult_cuda_kernel<<<gridSize, blockSize>>>(A->rows, d_row_ptr, d_col_indices, d_values, d_x, d_y);

    // registrazione del tempo di esecuzione
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Copia il risultato dalla GPU alla CPU
    hipMemcpy(y, d_y, A->rows * sizeof(double), hipMemcpyDeviceToHost);

    // allocazione del tempo
    hipEventElapsedTime(elapsed_time, start, stop);

    // Deallocazione della memoria sulla GPU
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_row_ptr);
    hipFree(d_col_indices);
    hipFree(d_values);
    hipFree(d_x);
    hipFree(d_y);
}


void select_and_run_cuda_csr(CSRMatrix *A, double *x, double *y, float *elapsed_time) {
    int avg_nnz_per_row = A->nnz / A->rows;
    double density = (double)A->nnz / A->rows;
    density /= A->cols;
    // Euristica per scegliere il kernel
    if (avg_nnz_per_row < 25 || density > 1.5e-3) {
        // Matrici piccole o con pochi elementi per riga -> Kernel row-based (più semplice)
        cuda_csr_mult(A, x, y, elapsed_time);
    } else {
        // Matrici grandi e con molte operazioni per riga -> Kernel warp-based
        cuda_csr_mult_warp(A, x, y, elapsed_time); // Funzione che esegue il primo kernel
    }
}
