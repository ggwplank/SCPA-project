#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "utils.h"

#define WARP_SIZE 32
#define BLOCK_SIZE 128  // Multiplo di 32

__global__ void csr_mult_cuda_kernel(int num_rows, int *d_row_ptr, int *d_col_indices, 
                                     double *d_values, double *d_x, double *d_y) {
    __shared__ double sdata[BLOCK_SIZE];  
    __shared__ int ptrs[BLOCK_SIZE / WARP_SIZE][2];

    const int thread_id = blockIdx.x * blockDim.x + threadIdx.x;  // ID globale del thread
    const int thread_in_warp = threadIdx.x & (WARP_SIZE - 1);     // ID del thread dentro il warp
    const int warp_id = thread_id / WARP_SIZE;                    // ID globale del warp
    const int warp_in_block = threadIdx.x / WARP_SIZE;            // Warp ID locale nel blocco
    const int num_warps = (BLOCK_SIZE / WARP_SIZE) * gridDim.x;   // Numero totale di warps

    // Ogni warp si occupa di una riga della matrice
    for (int row = warp_id; row < num_rows; row += num_warps) {
        // I primi due thread di ogni warp caricano gli indici di inizio/fine riga
        if (thread_in_warp < 2) {
            ptrs[warp_in_block][thread_in_warp] = d_row_ptr[row + thread_in_warp];
        }
        __syncthreads(); // Sincronizzazione per garantire che ptrs sia stato aggiornato

        int row_start = ptrs[warp_in_block][0];  
        int row_end = ptrs[warp_in_block][1];    

        // Calcolo del prodotto locale
        double sum = 0.0;
        for (int j = row_start + thread_in_warp; j < row_end; j += WARP_SIZE) {
            sum += d_values[j] * d_x[d_col_indices[j]];
        }

        // Riduzione all'interno del warp
        for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
            sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
        }

        // Il primo thread del warp scrive il risultato finale nella memoria globale
        if (thread_in_warp == 0) {
            d_y[row] = sum;
        }
    }
}


void cuda_csr_mult(CSRMatrix *A, double *x, double *y, float *elapsed_time) {
    int *d_row_ptr, *d_col_indices;
    double *d_values, *d_x, *d_y;

    // Allocazione memoria sulla GPU
    hipMalloc((void **)&d_row_ptr, (A->rows + 1) * sizeof(int));
    hipMalloc((void **)&d_col_indices, A->nnz * sizeof(int));
    hipMalloc((void **)&d_values, A->nnz * sizeof(double));
    hipMalloc((void **)&d_x, A->cols * sizeof(double));
    hipMalloc((void **)&d_y, A->rows * sizeof(double));

    // Copia dati dalla CPU alla GPU
    hipMemcpy(d_row_ptr, A->row_ptr, (A->rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_indices, A->col_indices, A->nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values, A->values, A->nnz * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, A->cols * sizeof(double), hipMemcpyHostToDevice);

    // Configurazione e lancio del kernel CUDA
    int threads_per_block = 128;  // Multiplo di 32 per i warp
    int num_warps = (A->rows + WARP_SIZE - 1) / WARP_SIZE;
    int num_blocks = (num_warps + (threads_per_block / WARP_SIZE) - 1) / (threads_per_block / WARP_SIZE);

    // Configurazione per il calcolo del tempo di esecuzione
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start, 0);

    csr_mult_cuda_kernel<<<num_blocks, threads_per_block>>>(A->rows, d_row_ptr, d_col_indices, d_values, d_x, d_y);


    // registrazione del tempo di esecuzione
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Copia il risultato dalla GPU alla CPU
    hipMemcpy(y, d_y, A->rows * sizeof(double), hipMemcpyDeviceToHost);

    // allocazione del tempo
    hipEventElapsedTime(elapsed_time, start, stop);

    // Deallocazione della memoria sulla GPU
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_row_ptr);
    hipFree(d_col_indices);
    hipFree(d_values);
    hipFree(d_x);
    hipFree(d_y);
}

