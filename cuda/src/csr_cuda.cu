#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "utils.h"

// Kernel CUDA per il prodotto matrice-vettore CSR
__global__ void csr_mult_cuda_kernel(int rows, int *d_row_ptr, int *d_col_indices, double *d_values, double *d_x, double *d_y) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;  // Ogni thread processa una riga

    if (row < rows) {
        double sum = 0.0;

        for (int j = d_row_ptr[row]; j < d_row_ptr[row + 1]; j++)
            sum += d_values[j] * d_x[d_col_indices[j]];
        
        d_y[row] = sum;
    }
}

void cuda_csr_mult(CSRMatrix *A, double *x, double *y, float *elapsed_time) {
    int *d_row_ptr, *d_col_indices;
    double *d_values, *d_x, *d_y;

    // Allocazione memoria sulla GPU
    hipMalloc((void **)&d_row_ptr, (A->rows + 1) * sizeof(int));
    hipMalloc((void **)&d_col_indices, A->nnz * sizeof(int));
    hipMalloc((void **)&d_values, A->nnz * sizeof(double));
    hipMalloc((void **)&d_x, A->cols * sizeof(double));
    hipMalloc((void **)&d_y, A->rows * sizeof(double));

    // Copia dati dalla CPU alla GPU
    hipMemcpy(d_row_ptr, A->row_ptr, (A->rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_indices, A->col_indices, A->nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_values, A->values, A->nnz * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, A->cols * sizeof(double), hipMemcpyHostToDevice);

    // Configurazione e lancio del kernel CUDA
    int blockSize = 256;
    int gridSize = (A->rows + blockSize - 1) / blockSize;

    // Configurazione per il calcolo del tempo di esecuzione
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    hipEventRecord(start, 0);

    csr_mult_cuda_kernel<<<gridSize, blockSize>>>(A->rows, d_row_ptr, d_col_indices, d_values, d_x, d_y);

    // registrazione del tempo di esecuzione
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Copia il risultato dalla GPU alla CPU
    hipMemcpy(y, d_y, A->rows * sizeof(double), hipMemcpyDeviceToHost);

    // allocazione del tempo
    hipEventElapsedTime(elapsed_time, start, stop);

    // Deallocazione della memoria sulla GPU
    hipEventDestroy(start);
    hipEventDestroy(stop);
    hipFree(d_row_ptr);
    hipFree(d_col_indices);
    hipFree(d_values);
    hipFree(d_x);
    hipFree(d_y);
}
